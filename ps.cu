#include "hip/hip_runtime.h"

#include<bits/stdc++.h>
#include <iostream>
#include <numeric>

using namespace std;

#define BLOCK_SIZE 1

__global__ void sum(int* input)  		 
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;      
	

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads)  
		{     
			const int fst = tid * step_size * 2;     
			const int snd = fst + step_size;	
			input[fst] += input[snd];				 
					
		}

		step_size <<= 1; 			
		number_of_threads >>= 1;	
	}
	
}

int main()
{
	cout <<"in main"<<endl;
	
	int count=0;
	// cout << "Enter the number of elements:\n" << endl;
	// cin>>count;
	count = 6;
	
	const int size = count * sizeof(int);
	
	int h[count];
	cout << "Enter the elements:\n" << endl;
	for(int i=0;i<count;i++)
	{
		// cout << "\n";
		// cin >> h[i];
		h[i] = i+1;
	}
	

	int* d;							//GPU parameter
       
	
	hipMalloc(&d, size);			
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
	sum <<<1, count/2 >>>(d);							// call kernel function <<<number of blocks, number of threads= number of elements/2
	
	int result;
	
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);	

	cout << "Sum is " << result << endl;

	getchar();

	hipFree(d);		


	return 0;
}
